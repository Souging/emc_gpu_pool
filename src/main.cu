#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdint.h>
#include <stdio.h>
#include <getopt.h>
#include <string.h>
#include "sha3.cuh"
#define BLOCKS 128
#define THREADS 256
#define N 100000000000  
__device__ int lock = 0;  
__device__ void lock_acquire() {
    while (atomicCAS(&lock, 0, 1) != 0) {
        __threadfence();  
    }
}
__device__ void lock_release() {
    atomicExch(&lock, 0);  
}
// Fixed constants (Host side only)
__constant__ uint8_t DIFFICULTY[4] = {0x22, 0x33, 0x99,0x99};
__constant__ uint8_t CURRENT_CHALLENGE[32] = {0x78, 0x90, 0x55, 0x58, 0x00, 0x90, 0x00, 0x90,
                                              0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00,
                                              0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00,
                                              0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00};
__constant__ uint8_t MINER_ADDRESS[20] = {
    0x32, 0xb2, 0xc3, 0xee, 0x46, 0xa6, 0x56, 0x65,
    0x28, 0x2d, 0xbd, 0x50, 0x32, 0x8a, 0xfa, 0x46,
    0x79, 0xec, 0x17, 0x86
};

// Device function to check if hash matches difficulty
__device__ char int_to_hex(int i) {
    if (i < 10) return '0' + i;
    return 'a' + (i - 10);
}

__device__ void byte_to_hex(uint8_t byte, char* output) {
    output[0] = int_to_hex((byte >> 4) & 0xF);
    output[1] = int_to_hex(byte & 0xF);
}

__global__ void calculate() {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    __shared__ uint8_t shared_difficulty[3];
    __shared__ uint8_t shared_challenge[32];
    __shared__ uint8_t shared_miner_address[20];
    if (threadIdx.x < 3) shared_difficulty[threadIdx.x] = DIFFICULTY[threadIdx.x];
    if (threadIdx.x < 32) shared_challenge[threadIdx.x] = CURRENT_CHALLENGE[threadIdx.x];
    if (threadIdx.x < 20) shared_miner_address[threadIdx.x] = MINER_ADDRESS[threadIdx.x];
    __syncthreads();
    hiprandState state;
    hiprand_init((unsigned long long)clock() + tid, 0, 0, &state);
    //uint8_t dev_difficulty[4] = {DIFFICULTY[0], DIFFICULTY[1], DIFFICULTY[2], DIFFICULTY[3]};
    //uint8_t local_challenge[32];
    //uint8_t local_miner_address[20];
    //memcpy(local_challenge, shared_challenge, 32);
    //memcpy(local_miner_address, shared_miner_address, 20);
    uint64_t block = (uint64_t)(hiprand_uniform_double(&state) * UINT64_MAX);
    uint8_t data[96];
    uint8_t hash[32];
    uint8_t blockBytes[32];
    memcpy(&data[32], shared_challenge, 32);
    memset(&data[64], 0, 12);
    memcpy(&data[76], shared_miner_address, 20);  
    for (uint64_t i = 0; i < N; i++) {
        
        for (int j = 0; j < 32; j++) {
            //blockBytes[j] = hiprand(&state) & 0xFF;
            ((uint32_t*)blockBytes)[j] = hiprand(&state);
            //((uint32_t*)blockBytes)[j] ^= (hiprand(&state) << 16);
            //blockBytes[j] ^= (hiprand(&state) << 16);

        }
        memcpy(data, blockBytes, 32);
        sha3_return_t ok = sha3_HashBuffer(256, SHA3_FLAGS_KECCAK, data, 96, hash, 32);
        if (ok != 0) {
            printf("bad params\n");
            return;
        }
        bool match = true;

        for(int j = 0; j<3; j++){
            if(hash[j] != shared_difficulty[j]){
                match = false;
                break;
            }
        }
        if (match) {
            lock_acquire();
            //char result1[65];  // 32 bytes * 2 chars per byte + 1 null terminator
            //for (int j = 0; j < 32; j++) {
            //    byte_to_hex(hash[j], &result1[j*2]);
            //}
            //result1[64] = '\0'; 
            //printf("thread => %d hash :0x%s\n", tid, result1);
            
            char result[65];  // 32 bytes * 2 chars per byte + 1 null terminator
            for (int j = 0; j < 32; j++) {
                byte_to_hex(blockBytes[j], &result[j*2]);
            }
            result[64] = '\0'; 
            printf("thread => %d randomValue :0x%s\n", tid, result);
            lock_release();
        }
    }
}

int main(int argc, char *argv[]) {
	int opt;
	int poolIndex = -1; 
    //0x32b2C3eE46A65665282DbD50328AFA4679eC1786
	uint8_t MINER_ADDRESS1[20] = {0x32, 0xb2, 0xc3, 0xee, 0x46, 0xa6, 0x56, 0x65,0x28, 0x2d, 0xbd, 0x50, 0x32, 0x8a, 0xfa, 0x46,0x79, 0xec, 0x17, 0x86};
	uint8_t MINER_ADDRESS2[20] = {0x9f, 0x07, 0xd1, 0x4c, 0x88, 0xeb, 0x4f, 0x11, 0x1c, 0x9f, 0x9c, 0xc7, 0x66, 0x39, 0xb7, 0xca, 0xeb, 0x62, 0x2b, 0x4b};
	uint8_t MINER_ADDRESS3[20] = {0xd3, 0x79, 0xed, 0x77, 0x39, 0x39, 0x67, 0x58, 0x7e, 0x54, 0xf5, 0x0a, 0x4e, 0xf6, 0x28, 0xda, 0x08, 0x41, 0x7c, 0xb5};
	uint8_t MINER_ADDRESS4[20] = {0x6f, 0xcd, 0x09, 0x8f, 0x3c, 0x21, 0x21, 0x9b, 0x57, 0x2a, 0x5a, 0x1a, 0xf1, 0xfc, 0x82, 0xd2, 0x3f, 0x89, 0x23, 0xd7};
	while ((opt = getopt(argc, argv, "p:")) != -1) {
        switch (opt) {
            case 'p':
                poolIndex = atoi(optarg); 
                break;
            default:
                fprintf(stderr, "Usage: %s -p <pool_index (1-4)>\n", argv[0]);
                exit(EXIT_FAILURE);
        }
    }
	if (poolIndex < 1 || poolIndex > 4) {
        fprintf(stderr, "Invalid pool index. Please provide a value between 1 and 4.\n");
        exit(EXIT_FAILURE);
    }
	uint8_t *selectedMinerAddress;
    switch (poolIndex) {
        case 1: selectedMinerAddress = MINER_ADDRESS1; break;
        case 2: selectedMinerAddress = MINER_ADDRESS2; break;
        case 3: selectedMinerAddress = MINER_ADDRESS3; break;
        case 4: selectedMinerAddress = MINER_ADDRESS4; break;
    }
    hipMemcpyToSymbol(HIP_SYMBOL(DIFFICULTY), "\x12\x34\x88", 3);
    hipMemcpyToSymbol(HIP_SYMBOL(CURRENT_CHALLENGE), "\x78\x90\x55\x58\x00\x90\x00\x90\x00\x00\x00\x00\x00\x00\x00\x00\x00\x00\x00\x00\x00\x00\x00\x00\x00\x00\x00\x00\x00\x00\x00\x00", 32);
    hipMemcpyToSymbol(HIP_SYMBOL(MINER_ADDRESS), selectedMinerAddress, 20);
    calculate<<<BLOCKS, THREADS>>>();
    hipDeviceSynchronize();
    exit(EXIT_SUCCESS);
}
